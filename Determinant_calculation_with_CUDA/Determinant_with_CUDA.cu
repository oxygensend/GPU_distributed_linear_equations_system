#include "hip/hip_runtime.h"
#include ""
#include <fstream>
#include <iostream>
#include <hip/hip_runtime.h>
//#include "hdf5.h"
#include <highfive/H5File.hpp>
__global__ void detGaussKernel(double* mat, int n, double* det) {
    int i, j, k;
    double ratio;

    // Forward elimination
    for (i = 0; i < n - 1; i++) {
        for (j = i + 1; j < n; j++) {
            ratio = mat[j * n + i] / mat[i * n + i];

            for (k = i + 1; k < n; k++) {
                mat[j * n + k] -= ratio * mat[i * n + k];
            }
        }
    }

    // Calculate determinant
    *det = 1.0;
    for (i = 0; i < n; i++) {
        *det *= mat[i * n + i];
    }
}

int main(int argc, char* argv[]) {
    int n = 100;
    double* mat, * d_mat;
    double det, * d_det;

    mat = new double[n * n];
    hipMalloc(&d_mat, n * n * sizeof(double));
    hipMalloc(&d_det, sizeof(double));
    std::fstream newfile;
    newfile.open(argv[1], std::ios::in);
    // Fill matrix with random values
    for (int i = 0; i < n * n; i++) {
        newfile >> mat[i];

    }

    // Copy matrix to device
    hipMemcpy(d_mat, mat, n * n * sizeof(double), hipMemcpyHostToDevice);

    // Calculate determinant on device
    detGaussKernel << <1, 1 >> > (d_mat, n, d_det);

    // Copy determinant from device
    hipMemcpy(&det, d_det, sizeof(double), hipMemcpyDeviceToHost);

    std::cout << "Determinant: " << det << std::endl;

    // Free memory
    delete[] mat;
    hipFree(d_mat);
    hipFree(d_det);

    return 0;
}
