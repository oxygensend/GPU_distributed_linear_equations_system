#include "hip/hip_runtime.h"
#include ""
#include <fstream>
#include <iostream>
#include <hip/hip_runtime.h>
//#include "hdf5.h"
#include <highfive/H5File.hpp>

__global__ void detGaussKernel(double* mat, int n, double* det) {
    int i, j, k;
    double ratio;

    // Forward elimination
    for (i = 0; i < n - 1; i++) {
        for (j = i + 1; j < n; j++) {
            ratio = mat[j * n + i] / mat[i * n + i];

            for (k = i + 1; k < n; k++) {
                mat[j * n + k] -= ratio * mat[i * n + k];
            }
        }
    }

    // Calculate determinant
    *det = 1.0;
    for (i = 0; i < n; i++) {
        *det *= mat[i * n + i];
    }
}

int main(int argc, char* argv[]) {
    double* d_mat;
    double det, * d_det;

    HighFive::File file(argv[1], HighFive::File::ReadOnly);

    auto dataset = file.getDataSet("dataset_1");
    auto dataspace = dataset.getSpace();
    std::vector<size_t> dims(dataspace.getDimensions());
    std::vector<double> data(dims[0] * dims[1]);
    dataset.read(data.data());

    double* a = &data[0];

    const int n = sqrt(data.size());
    hipMalloc(&d_mat, n * n * sizeof(double));
    hipMalloc(&d_det, sizeof(double));




    // Copy matrix to device
    hipMemcpy(d_mat, a, n * n * sizeof(double), hipMemcpyHostToDevice);

    // Calculate determinant on device
    detGaussKernel << <1, 1 >> > (a, n, d_det);

    // Copy determinant from device
    hipMemcpy(&det, d_det, sizeof(double), hipMemcpyDeviceToHost);

    std::cout << "Determinant: " << det << std::endl;

    // Free memory

    hipFree(d_mat);
    hipFree(d_det);

    return 0;
}
