#include "hip/hip_runtime.h"
#include ""
#include <fstream>
#include <iostream>
#include <hip/hip_runtime.h>

// Function to compute the determinant of a matrix using the Laplace expansion method
__device__ float determinantLaplace(float* matrix, int n) {
    if (n == 1) {
        return matrix[0];
    }

    float result = 0;
    float sign = 1;

    // Allocate memory for submatrix
    float* submatrix = new float[(n - 1) * (n - 1)];

    // Compute the determinant using the Laplace expansion method
    for (int i = 0; i < n; i++) {
        // Extract submatrix
        int sub_i = 0;
        for (int j = 1; j < n; j++) {
            int sub_j = 0;
            for (int k = 0; k < n; k++) {
                if (k != i) {
                    submatrix[sub_i * (n - 1) + sub_j] = matrix[j * n + k];
                    sub_j++;
                }
            }
            sub_i++;
        }

        // Compute the determinant of the submatrix
        float sub_det = determinantLaplace(submatrix, n - 1);

        // Add to result
        result += sign * matrix[i] * sub_det;
        sign = -sign;
    }

    // Free memory for submatrix
    delete[] submatrix;

    return result;
}

// Kernel function to compute the determinant of a matrix using the Laplace expansion method
__global__ void determinantLaplaceKernel(float* matrix, float* det, int n) {
    int tid = threadIdx.x;

    // Compute determinant using the Laplace expansion method
    det[tid] = determinantLaplace(matrix, n);
}

// Function to compute the determinant of a matrix using the Laplace expansion method
float determinantLaplaceCUDA(float* matrix, int n) {
    // Allocate device memory
    float* d_matrix, * d_det;
    hipMalloc(&d_matrix, n * n * sizeof(float));
    hipMalloc(&d_det, n * sizeof(float));

    // Copy matrix to device memory
    hipMemcpy(d_matrix, matrix, n * n * sizeof(float), hipMemcpyHostToDevice);

    // Launch kernel to compute determinant using the Laplace expansion method
    determinantLaplaceKernel << <1, n >> > (d_matrix, d_det, n);

    // Copy determinant from device to host memory
    float result;
    hipMemcpy(&result, d_det, sizeof(float), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_matrix);
    hipFree(d_det);

    return result;
}

int main(int argc, char* argv[]) {
    // Define matrix
    if (argc != 1) {
        perror("You must specify file name as first argument.");
        exit(-1);
    }

    std::fstream newfile;
    newfile.open("data0.txt", std::ios::in); //open a file to perform read operation using file object

    const int Nrows = 100;
    float h_A[Nrows*Nrows];

    for (int i = 0; i < Nrows*Nrows; i++){
            newfile >> h_A[i];
    }
    // Compute determinant
    float det = determinantLaplaceCUDA(h_A, Nrows);

    // Print result

    // Print result
    std::cout << "Determinant: " << det << std::endl;

    return 0;
}



