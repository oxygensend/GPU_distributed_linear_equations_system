#include "hip/hip_runtime.h"
#include ""
#include <fstream>
#include <iostream>
#include <hip/hip_runtime.h>
//#include "hdf5.h"
#include <highfive/H5File.hpp>

using HighFive::File;

__device__ double det3(int n, int w, int* WK, double** A)
{
    int    i, j, k, m, * KK;
    double s;

    if (n == 1)                     // sprawdzamy warunek zako�czenia rekurencji

        return A[w][WK[0]];    // macierz 1 x 1, wyznacznik r�wny elementowi

    else
    {

        KK = new int[n - 1];        // tworzymy dynamiczny wektor kolumn

        s = 0;                         // zerujemy warto�� rozwini�cia
        m = 1;                         // pocz�tkowy mno�nik

        for (i = 0; i < n; i++)       // p�tla obliczaj�ca rozwini�cie
        {

            k = 0;                       // tworzymy wektor kolumn dla rekurencji

            for (j = 0; j < n - 1; j++) // ma on o 1 kolumn� mniej ni� WK
            {
                if (k == i) k++;          // pomijamy bie��c� kolumn�
                KK[j] = WK[k++];     // pozosta�e kolumny przenosimy do KK
            }

            s += m * A[w][WK[i]] * det3(n - 1, w + 1, KK, A);

            m = -m;                      // kolejny mno�nik

        }

        delete[] KK;                 // usuwamy zb�dn� ju� tablic� dynamiczn�

        return s;                      // ustalamy warto�� funkcji

    }
}


// Function to compute the determinant of a matrix using the Laplace expansion method
__device__ double determinantLaplace(double* matrix, int n) {
    if (n == 1) {
        return matrix[0];
    }

    double result = 0;
    double sign = 1;

    // Allocate memory for submatrix
    double* submatrix = new double[(n - 1) * (n - 1)];

    // Compute the determinant using the Laplace expansion method
    for (int i = 0; i < n; i++) {
        // Extract submatrix
        int sub_i = 0;
        for (int j = 1; j < n; j++) {
            int sub_j = 0;
            for (int k = 0; k < n; k++) {
                if (k != i) {
                    submatrix[sub_i * (n - 1) + sub_j] = matrix[j * n + k];
                    sub_j++;
                }
            }
            sub_i++;
        }

        // Compute the determinant of the submatrix
        double sub_det = determinantLaplace(submatrix, n - 1);

        // Add to result
        result += sign * matrix[i] * sub_det;
        sign = -sign;
    }

    // Free memory for submatrix
    delete[] submatrix;

    return result;
}

// Kernel function to compute the determinant of a matrix using the Laplace expansion method
__global__ void determinantLaplaceKernel(int n, int w, int* WK, double** A, double* det) {
    int tid = threadIdx.x;

    // Compute determinant using the Laplace expansion method
    det[tid] = det3( n,w,WK,A);
}

// Function to compute the determinant of a matrix using the Laplace expansion method
double determinantLaplaceCUDA(int n, int w, int* WK, double** A) {
    // Allocate device memory
    double* d_matrix, * d_det;
    hipMalloc(&d_matrix, n * n * sizeof(double));
    hipMalloc(&d_det, n * sizeof(double));

    // Copy matrix to device memory
    hipMemcpy(d_matrix, A, n * n * sizeof(double), hipMemcpyHostToDevice);

    // Launch kernel to compute determinant using the Laplace expansion method
    determinantLaplaceKernel << <1, n >> > (n, w, WK, A,d_det);

    // Copy determinant from device to host memory
    double result;
    hipMemcpy(&result, d_det, sizeof(double), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_matrix);
    hipFree(d_det);

    return result;
}

int main(int argc, char* argv[]) {
    // Define matrix
    if (argc != 1) {
        perror("You must specify file name as first argument.");
        exit(-1);
    }
    
    std::fstream newfile;
    newfile.open("data0.txt", std::ios::in); //open a file to perform read operation using file object

    const int Nrows = 100;
    double** h_A;
    h_A = new double* [Nrows];

    for (int i = 0; i < Nrows; i++)
    {

        h_A[i] = new double[Nrows];    

        for (int j = 0; j < Nrows; j++)
            newfile >> h_A[i][j]; 

    }
    int* WK = new int[Nrows];              

    for (int i = 0; i < Nrows; i++)       
        WK[i] = i;

    std::vector<double> result;
    /*
    HighFive::File file("data37.h5", HighFive::File::ReadOnly);

    auto dataset = file.getDataSet("dataset_1");
    auto dataspace = dataset.getSpace();
    std::vector<size_t> dims(dataspace.getDimensions());
    std::vector<double> data(dims[0] * dims[1]);
    dataset.read(data.data());
 
    double* a = &data[0];
 */
    // Compute determinant
    double det = determinantLaplaceCUDA(Nrows, 0, WK, h_A);
    //double det2 = det(Nrows,0,WK,h_A);

    // Print result
    std::cout << "Determinant: " << det << std::endl;

    return 0;
}



