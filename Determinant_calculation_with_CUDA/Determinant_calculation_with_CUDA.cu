#include "hip/hip_runtime.h"
#include ""
#include <fstream>
#include <iostream>
#include <hip/hip_runtime.h>
//#include "hdf5.h"
#include <highfive/H5File.hpp>


using HighFive::File;




// Function to compute the determinant of a matrix using the Laplace expansion method
__device__ double determinantLaplace(double* matrix, int n) {
    if (n == 1) {
        return matrix[0];
    }

    double result = 0;
    double sign = 1;

    // Allocate memory for submatrix
    double* submatrix ;
    hipMalloc(&submatrix, (n - 1) * (n - 1)* sizeof(double));
    // Compute the determinant using the Laplace expansion method
    for (int i = 0; i < n; i++) {
        // Extract submatrix
        int sub_i = 0;
        for (int j = 1; j < n; j++) {
            int sub_j = 0;
            for (int k = 0; k < n; k++) {
                if (k != i) {
# if __CUDA_ARCH__>=200
                    printf("%f", submatrix[sub_i * (n - 1) + sub_j]);

#endif
                    
                    submatrix[sub_i * (n - 1) + sub_j] = matrix[j * n + k];
                    sub_j++;
                }
            }
            sub_i++;
        }

        // Compute the determinant of the submatrix
        double sub_det = determinantLaplace(submatrix, n - 1);

        // Add to result
        result += sign * matrix[i] * sub_det;
        sign = -sign;
    }

    // Free memory for submatrix
    delete[] submatrix;

    return result;
}

// Kernel function to compute the determinant of a matrix using the Laplace expansion method
__global__ void determinantLaplaceKernel(double* matrix, int n, double* det) {
    int tid = threadIdx.x;

    // Compute determinant using the Laplace expansion method
    *det = determinantLaplace(matrix,n);
    printf("%f", det);

}

// Function to compute the determinant of a matrix using the Laplace expansion method
double determinantLaplaceCUDA(double* matrix, int n) {
    // Allocate device memory
    double* d_matrix;
    double * d_det;
    hipMalloc(&d_matrix, n * n * sizeof(double));
    hipMalloc(&d_det, sizeof(double));

    // Copy matrix to device memory
    hipMemcpy(d_matrix, matrix, n * n * sizeof(double), hipMemcpyHostToDevice);
    int threadsPerBlock = 256;
    int blocksPerGrid =(n + threadsPerBlock - 1) / threadsPerBlock;
    // Launch kernel to compute determinant using the Laplace expansion method
    determinantLaplaceKernel << <1, 1 >> > (matrix,n,d_det);
 
    // Copy determinant from device to host memory
    double* result;
    hipMemcpy(&result, d_det, sizeof(double), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_matrix);
    hipFree(d_det);

    return  *result;
;
}

int mai2n(int argc, char* argv[]) {
    // Define matrix
    if (argc != 1) {
        perror("You must specify file name as first argument.");
        exit(-1);
    }
    
    std::fstream newfile;
    newfile.open("data0.txt", std::ios::in); //open a file to perform read operation using file object

    const int Nrows = 5;
    double h_A[Nrows*Nrows];
    

    for (int i = 0; i < Nrows * Nrows; i++) {
        newfile >> h_A[i];
    }


    std::vector<double> result;
    /*
    HighFive::File file("data37.h5", HighFive::File::ReadOnly);

    auto dataset = file.getDataSet("dataset_1");
    auto dataspace = dataset.getSpace();
    std::vector<size_t> dims(dataspace.getDimensions());
    std::vector<double> data(dims[0] * dims[1]);
    dataset.read(data.data());

    double* a = &data[0];
     */
    // Compute determinant
    double det = determinantLaplaceCUDA( h_A, Nrows);
    //double det2 = det(Nrows,0,WK,h_A);

    // Print result
    std::cout << "Determinant: " << det << std::endl;

    return 0;
} 



