#include "device_launch_paraMeters.h"
#include "hip/hip_runtime.h"
#include <vector>
#include<iostream>
#include<iomanip>
#include<stdlib.h>
#include<stdio.h>
#include<assert.h>
#include<ostream>

#include <hipsolver.h>
#include <hipblas.h>
#include <hip/hip_runtime_api.h>
#include <algorithm>
#include <cassert>
#include <cstdlib>
#include <functional>

#include "Utilities.cuh"


#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/iterator/permutation_iterator.h>
#include <thrust/functional.h>

#include <thrust/fill.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

#include <thrust/copy.h>




/*************************/
/* STRIDED RANGE FUNCTOR */
/*************************/
template <typename Iterator>
class strided_range
{
public:

    typedef typename thrust::iterator_difference<Iterator>::type difference_type;

    struct stride_functor : public thrust::unary_function<difference_type, difference_type>
    {
        difference_type stride;

        stride_functor(difference_type stride)
            : stride(stride) {}

        __host__ __device__
            difference_type operator()(const difference_type& i) const
        {
            return stride * i;
        }
    };

    typedef typename thrust::counting_iterator<difference_type>                   CountingIterator;
    typedef typename thrust::transform_iterator<stride_functor, CountingIterator> TransformIterator;
    typedef typename thrust::permutation_iterator<Iterator, TransformIterator>     PermutationIterator;

    // type of the strided_range iterator
    typedef PermutationIterator iterator;

    // construct strided_range for the range [first,last)
    strided_range(Iterator first, Iterator last, difference_type stride)
        : first(first), last(last), stride(stride) {}

    iterator begin(void) const
    {
        return PermutationIterator(first, TransformIterator(CountingIterator(0), stride_functor(stride)));
    }

    iterator end(void) const
    {
        return begin() + ((last - first) + (stride - 1)) / stride;
    }

protected:
    Iterator first;
    Iterator last;
    difference_type stride;
};

int main(void)
{


        const int Nrows = 5;
        const int STRIDE = Nrows + 1;
        srand(time(0));

        double h_A[Nrows][Nrows] = {
           { 2.,    -2.,    -2.,    -2.,    -2.,},
           {-2.,     4.,     0.,     0.,     0.,},
           {-2.,     0.,     6.,     2.,     2.,},
           {-2.,     0.,     2.,     8.,     4.,},
           {-2.,     0.,     2.,     4.,     10.,}
        };

        // --- Setting the device matrix and moving the host matrix to the device
        double* d_A;

        gpuErrchk(hipMalloc(&d_A, Nrows * Nrows * sizeof(double)));

        gpuErrchk(hipMemcpy(d_A, h_A, Nrows * Nrows * sizeof(double), hipMemcpyHostToDevice));

        // --- cuSOLVE input/output parameters/arrays
        int work_size = 0;
        int* devInfo;

        gpuErrchk(hipMalloc(&devInfo, sizeof(int)));

     
        // --- CUDA solver initialization
        hipsolverHandle_t solver_handle;
      
        hipsolverDnCreate(&solver_handle);
       
        // --- CUDA CHOLESKY initialization
        cusolveSafeCall(hipsolverDnDpotrf_bufferSize(solver_handle, HIPBLAS_FILL_MODE_LOWER, Nrows, d_A, Nrows, &work_size));
  
        // --- CUDA POTRF execution
        double* work;   gpuErrchk(hipMalloc(&work, work_size * sizeof(double)));

        cusolveSafeCall(hipsolverDnDpotrf(solver_handle, HIPBLAS_FILL_MODE_LOWER, Nrows, d_A, Nrows, work, work_size, devInfo));
  
        int devInfo_h = 0;  
        gpuErrchk(hipMemcpy(&devInfo_h, devInfo, sizeof(int), hipMemcpyDeviceToHost));

        if (devInfo_h != 0) std::cout << "Unsuccessful potrf execution\n\n";

        hipsolverDnDestroy(solver_handle);

        // --- Strided reduction of the elements of d_A: calculating the product of the diagonal of the Cholesky factorization  
        thrust::device_ptr<double> dev_ptr = thrust::device_pointer_cast(d_A);
   
        typedef thrust::device_vector<double>::iterator Iterator;
        strided_range<Iterator> pos(dev_ptr, dev_ptr + Nrows * Nrows, STRIDE);

        try {
        double det = thrust::reduce(pos.begin(), pos.end(), 1., thrust::maximum<double>());
        det = det * det;

    }
    catch (std::exception& e) {
        printf(e.what());
    }
}