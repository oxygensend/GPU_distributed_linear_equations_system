#include <stdio.h>
#include <assert.h>


#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include <hipsolver.h>
#include <hipblas.h>
#include <hipfft/hipfft.h>

#include "Utilities.cuh"

#define DEBUG

#define PI_R         3.14159265358979323846f

/*******************/
/* iDivUp FUNCTION */
/*******************/
//extern "C" int iDivUp(int a, int b){ return ((a % b) != 0) ? (a / b + 1) : (a / b); }
__host__ __device__ int iDivUp(int a, int b) { return ((a % b) != 0) ? (a / b + 1) : (a / b); }

/********************/
/* CUDA ERROR CHECK */
/********************/
// --- Credit to http://stackoverflow.com/questions/14038589/what-is-the-canonical-way-to-check-for-errors-using-the-cuda-runtime-api
void gpuAssert(hipError_t code, const char* file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) { exit(code); }
	}
}

extern "C" void gpuErrchk(hipError_t ans) { gpuAssert((ans), __FILE__, __LINE__); }

/**************************/
/* CUSOLVE ERROR CHECKING */
/**************************/
static const char* _cusolverGetErrorEnum(hipsolverStatus_t error)
{
	switch (error)
	{
	case HIPSOLVER_STATUS_SUCCESS:
		return "CUSOLVER_SUCCESS";

	case HIPSOLVER_STATUS_NOT_INITIALIZED:
		return "HIPSOLVER_STATUS_NOT_INITIALIZED";

	case HIPSOLVER_STATUS_ALLOC_FAILED:
		return "HIPSOLVER_STATUS_ALLOC_FAILED";

	case HIPSOLVER_STATUS_INVALID_VALUE:
		return "HIPSOLVER_STATUS_INVALID_VALUE";

	case HIPSOLVER_STATUS_ARCH_MISMATCH:
		return "HIPSOLVER_STATUS_ARCH_MISMATCH";

	case HIPSOLVER_STATUS_EXECUTION_FAILED:
		return "HIPSOLVER_STATUS_EXECUTION_FAILED";

	case HIPSOLVER_STATUS_INTERNAL_ERROR:
		return "HIPSOLVER_STATUS_INTERNAL_ERROR";

	case HIPSOLVER_STATUS_MATRIX_TYPE_NOT_SUPPORTED:
		return "HIPSOLVER_STATUS_MATRIX_TYPE_NOT_SUPPORTED";

	}

	return "<unknown>";
}

inline void __cusolveSafeCall(hipsolverStatus_t err, const char* file, const int line)
{
	if (HIPSOLVER_STATUS_SUCCESS != err) {
		fprintf(stderr, "CUSOLVE error in file '%s', line %d, error: %s \nterminating!\n", __FILE__, __LINE__, \
			_cusolverGetErrorEnum(err)); \
			assert(0); \
	}
}

extern "C" void cusolveSafeCall(hipsolverStatus_t err) { __cusolveSafeCall(err, __FILE__, __LINE__); }
