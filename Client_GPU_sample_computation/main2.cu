#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hipsolver.h>

int main() {
    const int n = 3; // size of the matrix
    const int nrhs = 1; // number of right-hand sides

    // example matrix - macierz zapisujemy {[0,0], [0,1], [0,2], [1,0], [1,1], [1,2] ,[2,0], [2,1], [2,2]}  -kolumnami
    float A[n*n] = {2, 4, -2, 1, -6, 7, 1, 0, 2};
    float B[n*nrhs] = {5,-2,9};

    // allocate memory on the device
    float *dA, *dB, *dX;
    int *dipiv; // pivoting information

    hipMalloc((void**)&dA, n * n * sizeof(float));
    hipMalloc((void**)&dB, n * nrhs * sizeof(float));
    hipMalloc((void**)&dX, n * nrhs * sizeof(float));
    hipMalloc((void**)&dipiv, n * sizeof(int));

    // copy input matrices to the device
    hipMemcpy(dA, A, n * n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dB, B, n * nrhs * sizeof(float), hipMemcpyHostToDevice);

    // create cusolverDn handle
    hipsolverHandle_t handle;
    hipsolverDnCreate(&handle);

    // factorize and solve the linear system
    int niter, dinfo;
    size_t workspace_size;
    hipsolverStatus_t status = hipsolverDnSSgesv_bufferSize(handle, n, nrhs, dA, n, dipiv, dB, n, dX, n, NULL, &workspace_size);

    if (status != HIPSOLVER_STATUS_SUCCESS) {
        printf("Error querying workspace size: %d\n", status);
        return 1;
    }

    status = hipsolverDnSSgesv(handle, n, nrhs, dA, n, dipiv, dB, n, dX, n, NULL, workspace_size, &niter, &dinfo);

    printf("%d %long %d %d\n", status, workspace_size, dinfo, niter);
    if (status != HIPSOLVER_STATUS_SUCCESS) {
        printf("Error solving linear system: %d\n", status);
        return 1;
    }

    if (dinfo != 0) {
        printf("Error solving linear system: dinfo = %d\n", dinfo);
        return 1;
    }

    // copy the solution back to the host
    float X[n*nrhs];
    hipMemcpy(X, dX, n * nrhs * sizeof(float), hipMemcpyDeviceToHost);

    // print the solution
    printf("Solution:\n");
    for (int i = 0; i < n; i++) {
        printf("%f\n", X[i]);
    }

    // free memory
    hipFree(dA);
    hipFree(dB);
    hipFree(dX);
    hipFree(dipiv);
    hipsolverDnDestroy(handle);

    return 0;
}
